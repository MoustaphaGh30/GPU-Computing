#include "hip/hip_runtime.h"
#include "common.h"

#include "timer.h"

__global__ void vecMax_kernel(double* a, double* b, double* c, unsigned int M) {

	int i=blockDim.x*blockIdx.x+threadIdx.x;
	
	if(i<M){
	c[i] = (a[i] > b[i])?a[i]:b[i];
	}

}

void vecMax_gpu(double* a, double* b, double* c, unsigned int M) {

    	Timer timer;

    	// Allocate GPU memory
    	startTime(&timer);
		double *a_d, *b_d, *c_d;
		hipMalloc((void **)&a_d, M*sizeof(double)); 
		hipMalloc((void **)&b_d, M*sizeof(double));
		hipMalloc((void **)&c_d, M*sizeof(double));

    	hipDeviceSynchronize();
    	stopTime(&timer);
    	printElapsedTime(timer, "Allocation time");

    	
	// Copy data to GPU
    	startTime(&timer);
		hipMemcpy(a_d, a, M*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(b_d, b, M*sizeof(double), hipMemcpyHostToDevice);
	
    	hipDeviceSynchronize();
    	stopTime(&timer);
    	printElapsedTime(timer, "Copy to GPU time");

    	
	// Call kernel
    	startTime(&timer);
		const unsigned int numThreadsPerBlock=1024;
		const unsigned int numBlocks=(M+numThreadsPerBlock-1)/numThreadsPerBlock;
		vecMax_kernel <<<numBlocks, numThreadsPerBlock>>>(a_d, b_d, c_d, M);

    	hipDeviceSynchronize();
    	stopTime(&timer);
    	printElapsedTime(timer, "Kernel time", GREEN);

    	
	// Copy data from GPU
    	startTime(&timer);
		hipMemcpy(c, c_d, M*sizeof(double), hipMemcpyDeviceToHost);

    	hipDeviceSynchronize();
    	stopTime(&timer);
    	printElapsedTime(timer, "Copy from GPU time");

    	
	// Free GPU memory
    	startTime(&timer);
		hipFree(a_d);
		hipFree(b_d);
		hipFree(c_d);

    	hipDeviceSynchronize();
    	stopTime(&timer);
    	printElapsedTime(timer, "Deallocation time");

}

